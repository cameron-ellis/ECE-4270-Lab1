
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add_matrix(float *d_out, float *d_a, float * d_b){
	int idx = threadIdx.x;
	float a = d_a[idx];
	float b = d_b[idx];
	d_out[idx] = a + b;
}

int main(int argc, char ** argv){
	const int ARRAY_SIZE = 4;
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);
	
	float h_in[ARRAY_SIZE];
	for (int i=0; i < ARRAY_SIZE; i++){
		h_in[i] = float(i);
	}
    float h_in2[ARRAY_SIZE];
	for (int i=0; i < ARRAY_SIZE; i++){
		h_in2[i] = float(i + 4);
	}
	float h_out[ARRAY_SIZE];
    printf("Input matrix 1:\n");
    for (int i =0; i<ARRAY_SIZE; i++){
		printf("%.2f", h_in[i]);
		printf(((i % 2) !=1)? "\t": "\n");
	}
    printf("\nInput matrix 2:\n");
    for (int i =0; i<ARRAY_SIZE; i++){
		printf("%.2f", h_in2[i]);
		printf(((i % 2) !=1)? "\t": "\n");
	}

	float *d_in;
	float *d_in2;
	float *d_out;

	hipMalloc((void**) &d_in, ARRAY_BYTES); 
	hipMalloc((void**) &d_in2, ARRAY_BYTES); 
	hipMalloc((void**) &d_out, ARRAY_BYTES); 

	hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(d_in2, h_in2, ARRAY_BYTES, hipMemcpyHostToDevice);

	add_matrix<<<1, ARRAY_SIZE>>>(d_out, d_in, d_in2);

	hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

    printf("\nOutput matrix:\n");
	for (int i =0; i<ARRAY_SIZE; i++){
		printf("%.2f", h_out[i]);
		printf(((i % 2) !=1)? "\t": "\n");
	}
	
	hipFree(d_in);
	hipFree(d_in2);
	hipFree(d_out);

	return 0;
}